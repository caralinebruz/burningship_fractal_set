#include "hip/hip_runtime.h"
#define _GLIBCXX_USE_CXX11_ABI 0

#include <iostream>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <opencv2/opencv.hpp>

#include "utils.h"

using namespace std;
using namespace cv;

namespace
{
  //! [escape-time-algorithm]
  // pixels that should be included in the set take a lot of iterations
  // pixels exluded from the set take few iterations ie. they "escape" quickly
  __device__ int burningship(const float &cr, const float &ci, const int max)
  {
      float zr = 0;
      float zi = 0;
      float re = 0;
      float im = 0;

      for (int t = 0; t < max; t++)
      {
          if ((zr * zr + zi * zi) > 4.0f) {
              return t;
          }

          // z = abs(z*z) + c;
          re = zr * zr - zi * zi + cr;
          im = fabs(zr * zi) * 2.0 + ci;

          zr = re;
          zi = im;
      }
      return max;
    }

  //! [burningship-grayscale-value]
  // converts the number of iterations taken, to be a grayscale value
  __device__ int burningshipFormula(const float &cr, const float &ci, const int maxIter=500) {

      int value = burningship(cr, ci, maxIter);

      if(maxIter - value == 0)
      {
          return 0;
      }

      int grayscale_val = ::roundf(sqrt(value / (float) maxIter) * 255);
      return grayscale_val;
  }

  __global__ void parallelburningship(int *pixelMatrix, const int rows, const int cols, const float x1, const float y1, const float scaleX, const float scaleY, const int maxIter)
  {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= rows || j >= cols) {
        return;
    }

    float x0 = j / scaleX + x1;
    float y0 = i / scaleY + y1;

    float cr = x0;
    float ci = y0;

    int grayscale_value = burningship(cr, ci, maxIter);

    pixelMatrix[i + j * rows] = grayscale_value;
  }

  //! [burningship-sequential]
  __global__ void sequentialburningship(int* pixelMatrix, int rows, int cols, const float x1, const float y1, const float scaleX, const float scaleY)
  {
    for (int i = 0; i < rows; i++)
    {
      for (int j = 0; j < cols; j++)
      {
        // for each pixel in our image, figure out what coords that pixel
        // corresponds to in the domain of our problem
        float x0 = j / scaleX + x1;
        float y0 = i / scaleY + y1;

        // real is the x-axis
        float cr = x0;
        // imaginary is the y-axis
        float ci = y0;

        // get the grayscale value
        int grayscale_value = burningshipFormula(cr,ci);
        // counter intuitive, because this would normally be i*rows + j
        // but the true fractal is actually upside down, so we flip it
        // and make it i+rows*j
        pixelMatrix[i+j*rows] = grayscale_value;
      }

      if (i%200 == 0){
        printf("row %d/%d \n", i, rows);
      }
    }
  }

  void write_pixels_to_image_file(cv::Mat &img, int*pixelMatrix, int rows, int cols) {
  // uses openCV Mat datatype to write the pixel values and save image to disk
  for (int i = 0; i < rows; i++)
  {
    for (int j = 0; j < cols; j++)
    {
      int grayscale_int = pixelMatrix[i+j*rows];
      uchar value = (uchar) grayscale_int;
      img.ptr<uchar>(i)[j] = value;
    }
  }    
}
}

int main(int argc, char** argv)
{
  // define the image dimensions
  int rows_x = 9600; int cols_y = 10800;
  int maxIter = 500;

  Timer t;
  t.tic();

  int* pixelMatrix = (int*) malloc(rows_x * cols_y * sizeof(int));
  int* pixelMatrix_out = (int*) malloc(rows_x * cols_y * sizeof(int));
  int* d_pixelMatrix;

  float x1 = -2.2f, x2 = 2.2f;
  float y1 = -2.2f, y2 = 2.2f;

  float scaleX = cols_y / (x2 - x1); // ->  9600 / (2.2 - -2.2) ~= 2000
  float scaleY = rows_x / (y2 - y1); // ->  10800 / (2.2 - -2.2) ~= 2000

  //! [color the set of pixels in the set vs not in the set]
  t.tic();
  sequentialburningship<<<1,1>>>(pixelMatrix, rows_x, cols_y, x1, y1, scaleX, scaleY);
  printf("time to compute basic version = %f s\n", t.toc());

  // Render results to image file with openCV
  Mat burningshipImgSequential(rows_x, cols_y, CV_8U);
  write_pixels_to_image_file(burningshipImgSequential, pixelMatrix, rows_x, cols_y);
  imwrite("burningship.png", burningshipImgSequential);

  // Allocate device memory for pixelMatrix
  hipMalloc((void**)&d_pixelMatrix, sizeof(int) * rows_x * cols_y);

  // Transfer data from host to device memory
  hipMemcpy(d_pixelMatrix, pixelMatrix, sizeof(float) * rows_x * cols_y, hipMemcpyHostToDevice);
  printf("time to copy memory to device = %f s\n", t.toc());

  t.tic();
  // Run the burningship algorithm on the GPU
  parallelburningship<<<1, 1>>>(d_pixelMatrix, rows_x, cols_y, x1, y1, scaleX, scaleY, maxIter);
  hipDeviceSynchronize();
  printf("gpu execution = %f s\n", t.toc());

  // Transfer data back to host memory
  hipMemcpy(pixelMatrix_out, d_pixelMatrix, sizeof(float) * rows_x * cols_y, hipMemcpyDeviceToHost);

  // Convert pixel matrix to OpenCV Mat
  Mat img(rows_x, cols_y, CV_8UC1);
  write_pixels_to_image_file(img, pixelMatrix_out, rows_x, cols_y);

  // Save image to disk
  imwrite("burningship_gpu.png", img);

  // Free memory
  delete[] pixelMatrix;
  delete[] pixelMatrix_out;
  hipFree(d_pixelMatrix);

  return 0;
}
